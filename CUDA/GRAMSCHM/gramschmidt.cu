#include "hip/hip_runtime.h"
/**
 * gramschmidt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size */
#define M 2048
#define N 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif


void init_array(DATA_TYPE* A)
{
	int i, j;

	for (i = 0; i < M; i++)
	{
		for (j = 0; j < N; j++)
		{
			A[i*N + j] = ((DATA_TYPE) (i+1)*(j+1)) / (M+1);
		}
	}
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	hipSetDevice( GPU_DEVICE );	
	return;
}


__global__ void gramschmidt_kernel1(DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid==0)
	{
		DATA_TYPE nrm = 0.0;
		int i;
		for (i = 0; i < M; i++)
		{
			nrm += a[i * N + k] * a[i * N + k];
		}
      		r[k * N + k] = sqrt(nrm);
	}
}


__global__ void gramschmidt_kernel2(DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < M)
	{	
		q[i * N + k] = a[i * N + k] / r[k * N + k];
	}
}


__global__ void gramschmidt_kernel3(DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((j > k) && (j < N))
	{
		r[k*N + j] = 0.0;

		int i;
		for (i = 0; i < M; i++)
		{
			r[k*N + j] += q[i*N + k] * a[i*N + j];
		}
		
		for (i = 0; i < M; i++)
		{
			a[i*N + j] -= q[i*N + k] * r[k*N + j];
		}
	}
}


void gramschmidtCuda(DATA_TYPE* A, DATA_TYPE* R, DATA_TYPE* Q)
{
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 gridKernel1(1, 1);
	dim3 gridKernel2((size_t)ceil(((float)N) / ((float)DIM_THREAD_BLOCK_X)), 1);
	dim3 gridKernel3((size_t)ceil(((float)N) / ((float)DIM_THREAD_BLOCK_X)), 1);
	
	DATA_TYPE *A_gpu;
	DATA_TYPE *R_gpu;
	DATA_TYPE *Q_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * M * N);
	hipMalloc((void **)&R_gpu, sizeof(DATA_TYPE) * M * N);
	hipMalloc((void **)&Q_gpu, sizeof(DATA_TYPE) * M * N);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * M * N, hipMemcpyHostToDevice);
	
	int k;
	for (k = 0; k < N; k++)
	{
		gramschmidt_kernel1<<<gridKernel1,block>>>(A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
		gramschmidt_kernel2<<<gridKernel2,block>>>(A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
		gramschmidt_kernel3<<<gridKernel3,block>>>(A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
	}
	
	hipMemcpy(A, A_gpu, sizeof(DATA_TYPE) * M * N, hipMemcpyDeviceToHost);    
	hipMemcpy(R, R_gpu, sizeof(DATA_TYPE) * M * N, hipMemcpyDeviceToHost);
	hipMemcpy(Q, Q_gpu, sizeof(DATA_TYPE) * M * N, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(R_gpu);
	hipFree(Q_gpu);
}


int main(int argc, char *argv[])
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* R;
	DATA_TYPE* Q;
	
	A = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));
	R = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));  
	Q = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));  
	
	init_array(A);
	
	GPU_argv_init();
	
	t_start = rtclock();
	gramschmidtCuda(A, R, Q);
	t_end = rtclock();

#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lfs\n", t_end - t_start);
#endif
	
	free(A);
	free(R);
	free(Q);  

    	return 0;
}

