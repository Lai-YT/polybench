#include "hip/hip_runtime.h"
/**
 * gesummv.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size */
#define N 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 43532.0f
#define BETA 12313.0f

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif


void init(DATA_TYPE* A, DATA_TYPE* x)
{
  	int i, j;

 	for (i = 0; i < N; i++)
    {
    	x[i] = ((DATA_TYPE) i) / N;
      	
		for (j = 0; j < N; j++) 
		{
			A[i*N + j] = ((DATA_TYPE) i*j) / N;
		}
    }
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gesummv_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j = 0; j < N; j++)
		{	
			tmp[i] += a[i * N + j] * x[j];
			y[i] += b[i * N + j] * x[j];
		}
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}

void gesummvCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * N);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), 1);


	gesummv_kernel<<< grid, block>>>(A_gpu,B_gpu,x_gpu, y_gpu, tmp_gpu);
	hipDeviceSynchronize();

	hipMemcpy(tmp, tmp_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
	hipMemcpy(y, y_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(tmp_gpu);
}


int main(int argc, char *argv[])
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* x;  
	DATA_TYPE* y;
	DATA_TYPE* tmp;
	
	A = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
	x = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE)); 
	y = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

	init(A, x);
	
	GPU_argv_init();
	
	t_start = rtclock();
	gesummvCuda(A, B, x, y, tmp);
	t_end = rtclock();
#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lfs\n", t_end - t_start);
#endif
	
	free(A);
	free(B);  
	free(x);  
	free(y);
	free(tmp);

	return 0;
}

