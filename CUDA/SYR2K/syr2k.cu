#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size */
#define N 2048
#define M 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 12435
#define BETA 4546

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

#define cudaCheckReturn(ret) \
	do { \
		hipError_t cudaCheckReturn_e = (ret); \
		if (cudaCheckReturn_e != hipSuccess) { \
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
			fflush(stderr); \
		} \
		assert(cudaCheckReturn_e == hipSuccess); \
	} while(0)

#define cudaCheckKernel() \
	do { \
		cudaCheckReturn(hipGetLastError()); \
	} while(0)

void init_arrays(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j;
  
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			C[i*N + j] = ((DATA_TYPE) i*j + 2) / N;
		}

		for (j = 0; j < M; j++)
		{
			A[i*N + j] = ((DATA_TYPE) i*j) / N;
			B[i*N + j] = ((DATA_TYPE) i*j + 1) / N;
		}
	}
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	cudaCheckReturn(hipGetDeviceProperties(&deviceProp, GPU_DEVICE));
	cudaCheckReturn(hipSetDevice(GPU_DEVICE));
}


__global__ void syr2k_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < N) && (j < N))
	{
		c[i * N + j] *= BETA;
		
		int k;
		for(k = 0; k < M; k++)
		{
			c[i * N + j] += ALPHA * a[i * M + k] * b[j * M + k] + ALPHA * b[i * M + k] * a[j * M + k];
		}
	}
}


void syr2kCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C)
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	cudaCheckReturn(hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * M));
	cudaCheckReturn(hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * N * M));
	cudaCheckReturn(hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * N * N));
	cudaCheckReturn(hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice));
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil( ((float)N) / ((float)DIM_THREAD_BLOCK_X) ), (size_t)(ceil( ((float)N) / ((float)DIM_THREAD_BLOCK_Y) )));
	
	syr2k_kernel<<<grid,block>>>(A_gpu,B_gpu,C_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	
	cudaCheckReturn(hipMemcpy(C, C_gpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost));

	cudaCheckReturn(hipFree(A_gpu));
	cudaCheckReturn(hipFree(B_gpu));
	cudaCheckReturn(hipFree(C_gpu));
}


int main()
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* C;

	A = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));

	init_arrays(A, B, C);
    
	GPU_argv_init();
	
	t_start = rtclock();
	syr2kCuda(A, B, C);
	t_end = rtclock();
#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lf\n", t_end - t_start);
#endif

	free(A);
	free(B);
	free(C);

  	return 0;
}

