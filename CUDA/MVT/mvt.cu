#include "hip/hip_runtime.h"
/**
 * mvt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size */
#define N 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

#define cudaCheckReturn(ret) \
	do { \
		hipError_t cudaCheckReturn_e = (ret); \
		if (cudaCheckReturn_e != hipSuccess) { \
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
			fflush(stderr); \
		} \
		assert(cudaCheckReturn_e == hipSuccess); \
	} while(0)

#define cudaCheckKernel() \
	do { \
		cudaCheckReturn(hipGetLastError()); \
	} while(0)

void init_array(DATA_TYPE* A, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	int i, j;

	for (i = 0; i < N; i++)
	{
		x1[i] = ((DATA_TYPE) i) / N;
		x2[i] = ((DATA_TYPE) i + 1) / N;
		y1[i] = ((DATA_TYPE) i + 3) / N;
		y2[i] = ((DATA_TYPE) i + 4) / N;
		for (j = 0; j < N; j++)
		{
			A[i*N + j] = ((DATA_TYPE) i*j) / N;
		}
	}
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	cudaCheckReturn(hipGetDeviceProperties(&deviceProp, GPU_DEVICE));
	cudaCheckReturn(hipSetDevice(GPU_DEVICE));
}


__global__ void mvt_kernel1(DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j=0; j < N; j++)
		{
			x1[i] += a[i * N + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2(DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j=0; j < N; j++)
		{
			x2[i] += a[j * N + i] * y_2[j];	
		}
	}
}

void mvtCuda(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y_1, DATA_TYPE* y_2)
{
	DATA_TYPE* a_gpu;
	DATA_TYPE* x1_gpu;
	DATA_TYPE* x2_gpu;
	DATA_TYPE* y_1_gpu;
	DATA_TYPE* y_2_gpu;

	cudaCheckReturn(hipMalloc((void **)&a_gpu, sizeof(DATA_TYPE) * N * N));
	cudaCheckReturn(hipMalloc((void **)&x1_gpu, sizeof(DATA_TYPE) * N));
	cudaCheckReturn(hipMalloc((void **)&x2_gpu, sizeof(DATA_TYPE) * N));
	cudaCheckReturn(hipMalloc((void **)&y_1_gpu, sizeof(DATA_TYPE) * N));
	cudaCheckReturn(hipMalloc((void **)&y_2_gpu, sizeof(DATA_TYPE) * N));
	cudaCheckReturn(hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(x1_gpu, x1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(x2_gpu, x2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil((float)N/ ((float)DIM_THREAD_BLOCK_X)), 1);
	
	mvt_kernel1<<<grid,block>>>(a_gpu,x1_gpu,y_1_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	mvt_kernel2<<<grid,block>>>(a_gpu,x2_gpu,y_2_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());

	cudaCheckReturn(hipMemcpy(x1, x1_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));
	cudaCheckReturn(hipMemcpy(x2, x2_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));    
	
	cudaCheckReturn(hipFree(a_gpu));
	cudaCheckReturn(hipFree(x1_gpu));
	cudaCheckReturn(hipFree(x2_gpu));
	cudaCheckReturn(hipFree(y_1_gpu));
	cudaCheckReturn(hipFree(y_2_gpu));
}


int main()
{
	double t_start, t_end;

	DATA_TYPE* a;
	DATA_TYPE* x1;
	DATA_TYPE* x2;
	DATA_TYPE* y_1;
	DATA_TYPE* y_2;

	a = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
	x1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

	init_array(a, x1, x2, y_1, y_2);
	
	GPU_argv_init();

	t_start = rtclock();
	mvtCuda(a, x1, x2, y_1, y_2);
	t_end = rtclock();
#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lf\n", t_end - t_start);
#endif
	
	free(a);
	free(x1);
	free(x2);
	free(y_1);
	free(y_2);

  	return 0;
}

