#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size. */
#define NX 4096
#define NY 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

#define cudaCheckReturn(ret) \
	do { \
		hipError_t cudaCheckReturn_e = (ret); \
		if (cudaCheckReturn_e != hipSuccess) { \
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
			fflush(stderr); \
		} \
		assert(cudaCheckReturn_e == hipSuccess); \
	} while(0)

#define cudaCheckKernel() \
	do { \
		cudaCheckReturn(hipGetLastError()); \
	} while(0)


void init_array(DATA_TYPE *x, DATA_TYPE *A)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*(j)) / NX;
		}
	}
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	cudaCheckReturn(hipGetDeviceProperties(&deviceProp, GPU_DEVICE));
	cudaCheckReturn(hipSetDevice(GPU_DEVICE));
}

__global__ void atax_kernel1(DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NX)
	{
		int j;
		for(j=0; j < NY; j++)
		{
			tmp[i] += A[i * NY + j] * x[j];
		}
	}
}

__global__ void atax_kernel2(DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < NY)
	{
		int i;
		for(i=0; i < NX; i++)
		{
			y[j] += A[i * NY + j] * tmp[i];
		}
	}
}

void ataxGpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	cudaCheckReturn(hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY));
	cudaCheckReturn(hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NY));
	cudaCheckReturn(hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NY));
	cudaCheckReturn(hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NX));
	
	cudaCheckReturn(hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice));
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);

	atax_kernel1<<< grid1, block >>>(A_gpu,x_gpu,tmp_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	atax_kernel2<<< grid2, block >>>(A_gpu,y_gpu,tmp_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	
	cudaCheckReturn(hipMemcpy(tmp, tmp_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost));
	cudaCheckReturn(hipMemcpy(y, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost));

	cudaCheckReturn(hipFree(A_gpu));
	cudaCheckReturn(hipFree(x_gpu));
	cudaCheckReturn(hipFree(y_gpu));
	cudaCheckReturn(hipFree(tmp_gpu));
}

int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* x;
	DATA_TYPE* y;
	DATA_TYPE* tmp;

	A = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
	x = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));

	init_array(x, A);

	GPU_argv_init();
	
	t_start = rtclock();
	ataxGpu(A, x, y, tmp);
	t_end = rtclock();

#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lf\n", t_end - t_start);
#endif

	free(A);
	free(x);
	free(y);
	free(tmp);

  	return 0;
}

