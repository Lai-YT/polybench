#include "hip/hip_runtime.h"
/**
 * correlation.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size */
#define M 2048
#define N 2048

/* Thread block dimensions for kernel 1*/
#define DIM_THREAD_BLOCK_KERNEL_1_X 256
#define DIM_THREAD_BLOCK_KERNEL_1_Y 1

/* Thread block dimensions for kernel 2*/
#define DIM_THREAD_BLOCK_KERNEL_2_X 256
#define DIM_THREAD_BLOCK_KERNEL_2_Y 1

/* Thread block dimensions for kernel 3*/
#define DIM_THREAD_BLOCK_KERNEL_3_X 32
#define DIM_THREAD_BLOCK_KERNEL_3_Y 8

/* Thread block dimensions for kernel 4*/
#define DIM_THREAD_BLOCK_KERNEL_4_X 256
#define DIM_THREAD_BLOCK_KERNEL_4_Y 1

#define sqrt_of_array_cell(x,j) sqrt(x[j])

#define FLOAT_N 3214212.01f
#define EPS 0.005f

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

#define cudaCheckReturn(ret) \
	do { \
		hipError_t cudaCheckReturn_e = (ret); \
		if (cudaCheckReturn_e != hipSuccess) { \
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
			fflush(stderr); \
		} \
		assert(cudaCheckReturn_e == hipSuccess); \
	} while(0)

#define cudaCheckKernel() \
	do { \
		cudaCheckReturn(hipGetLastError()); \
	} while(0)

void init_arrays(DATA_TYPE* data)
{
	int i, j;
	
	for (i=0; i < (M+1); i++) 
	{
		for (j=0; j< (N+1); j++) 
		{
			data[i*(N+1) + j] = ((DATA_TYPE) i*j)/ (M+1);	
		}
	}
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	cudaCheckReturn(hipGetDeviceProperties(&deviceProp, GPU_DEVICE));
	cudaCheckReturn(hipSetDevice(GPU_DEVICE));
}

__global__ void mean_kernel(DATA_TYPE *mean, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if ((j >= 1) && (j < (M+1)))
	{
		mean[j] = 0.0;

		int i;
		for(i=1; i < (N+1); i++)
		{
			mean[j] += data[i*(M+1) + j];
		}
		
		mean[j] /= (DATA_TYPE)FLOAT_N;
	}
}

__global__ void std_kernel(DATA_TYPE *mean, DATA_TYPE *std, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	
	if ((j >= 1) && (j < (M+1)))
	{
		std[j] = 0.0;

		int i;
		for(i = 1; i < (N+1); i++)
		{
			std[j] += (data[i*(M+1) + j] - mean[j]) * (data[i*(M+1) + j] - mean[j]);
		}
		std[j] /= (FLOAT_N);
		std[j] = sqrt(std[j]);
		if(std[j] <= EPS) 
		{
			std[j] = 1.0;
		}
	}
}

__global__ void reduce_kernel(DATA_TYPE *mean, DATA_TYPE *std, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
	
	if ((i >= 1) && (i < (N+1)) && (j >= 1) && (j < (M+1)))
	{
		data[i*(M+1) + j] -= mean[j];
		data[i*(M+1) + j] /= (sqrt(FLOAT_N) * std[j]);
	}
}

__global__ void corr_kernel(DATA_TYPE *symmat, DATA_TYPE *data)
{
	int j1 = blockIdx.x * blockDim.x + threadIdx.x + 1;

	int i, j2;
	if ((j1 >= 1) && (j1 < M))
	{
		symmat[j1*(M+1) + j1] = 1.0;

		for (j2 = (j1 + 1); j2 < (M+1); j2++)
		{
			symmat[j1*(M+1) + j2] = 0.0;

			for(i = 1; i < (N+1); i++)
			{
				symmat[j1*(M+1) + j2] += data[i*(M+1) + j1] * data[i*(M+1) + j2];
			}
			symmat[j2*(M+1) + j1] = symmat[j1*(M+1) + j2];
		}
	}
}

void correlationCuda(DATA_TYPE* data, DATA_TYPE* mean, DATA_TYPE* stddev, DATA_TYPE* symmat)
{
	DATA_TYPE *data_gpu;
	DATA_TYPE *stddev_gpu;
	DATA_TYPE *mean_gpu;
	DATA_TYPE *symmat_gpu;

	cudaCheckReturn(hipMalloc((void **)&data_gpu, sizeof(DATA_TYPE) * (M+1) * (N+1)));
	cudaCheckReturn(hipMalloc((void **)&symmat_gpu, sizeof(DATA_TYPE) * (M+1) * (N+1)));
	cudaCheckReturn(hipMalloc((void **)&stddev_gpu, sizeof(DATA_TYPE) * (M+1)));
	cudaCheckReturn(hipMalloc((void **)&mean_gpu, sizeof(DATA_TYPE) * (M+1)));
	cudaCheckReturn(hipMemcpy(data_gpu, data, sizeof(DATA_TYPE) * (M+1) * (N+1), hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(symmat_gpu, symmat, sizeof(DATA_TYPE) * (M+1) * (N+1), hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(stddev_gpu, stddev, sizeof(DATA_TYPE) * (M+1), hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(mean_gpu, mean, sizeof(DATA_TYPE) * (M+1), hipMemcpyHostToDevice));
		
	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 grid1((size_t)(ceil((float)(M)) / ((float)DIM_THREAD_BLOCK_KERNEL_1_X)), 1);
	
	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid2((size_t)(ceil((float)(M)) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)), 1);
	
	dim3 block3(DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
	dim3 grid3((size_t)(ceil((float)(M)) / ((float)DIM_THREAD_BLOCK_KERNEL_3_X)), (size_t)(ceil((float)(N)) / ((float)DIM_THREAD_BLOCK_KERNEL_3_Y)));
	
	dim3 block4(DIM_THREAD_BLOCK_KERNEL_4_X, DIM_THREAD_BLOCK_KERNEL_4_Y);
	dim3 grid4((size_t)(ceil((float)(M)) / ((float)DIM_THREAD_BLOCK_KERNEL_4_X)), 1);

	mean_kernel<<< grid1, block1 >>>(mean_gpu, data_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	std_kernel<<< grid2, block2 >>>(mean_gpu, stddev_gpu, data_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	reduce_kernel<<< grid3, block3 >>>(mean_gpu, stddev_gpu, data_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	corr_kernel<<< grid4, block4 >>>(symmat_gpu, data_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());

	DATA_TYPE valueAtSymmatIndexMTimesMPlus1PlusMPoint = 1.0;
	cudaCheckReturn(hipMemcpy(&(symmat_gpu[(M)*(M+1) + (M)]), &valueAtSymmatIndexMTimesMPlus1PlusMPoint, sizeof(DATA_TYPE), hipMemcpyHostToDevice));

	cudaCheckReturn(hipMemcpy(data, data_gpu, sizeof(DATA_TYPE) * (M+1) * (N+1), hipMemcpyDeviceToHost));
	cudaCheckReturn(hipMemcpy(mean, mean_gpu, sizeof(DATA_TYPE) * (M+1), hipMemcpyDeviceToHost));
	cudaCheckReturn(hipMemcpy(stddev, stddev_gpu, sizeof(DATA_TYPE) * (M+1), hipMemcpyDeviceToHost));
	cudaCheckReturn(hipMemcpy(symmat, symmat_gpu, sizeof(DATA_TYPE) * (M+1) * (N+1), hipMemcpyDeviceToHost));
	
	cudaCheckReturn(hipFree(data_gpu));
	cudaCheckReturn(hipFree(symmat_gpu));
	cudaCheckReturn(hipFree(stddev_gpu));
	cudaCheckReturn(hipFree(mean_gpu));
}

int main()
{
	double t_start, t_end;

	DATA_TYPE* data;
	DATA_TYPE* mean;
	DATA_TYPE* stddev;
	DATA_TYPE* symmat;

	data = (DATA_TYPE*)malloc((M+1)*(N+1)*sizeof(DATA_TYPE));
	mean = (DATA_TYPE*)malloc((M+1)*sizeof(DATA_TYPE));
	stddev = (DATA_TYPE*)malloc((M+1)*sizeof(DATA_TYPE));
	symmat = (DATA_TYPE*)malloc((M+1)*(N+1)*sizeof(DATA_TYPE));

	init_arrays(data);
    
	GPU_argv_init();

	t_start = rtclock();
	correlationCuda(data, mean, stddev, symmat);
	t_end = rtclock();

#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lf\n", t_end - t_start);
#endif
    
	free(data);
	free(mean);
	free(stddev);
	free(symmat);

  	return 0;
}

