#include "hip/hip_runtime.h"
/**
 * bicg.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

/* Problem size. */
#define NX 4096
#define NY 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

#define cudaCheckReturn(ret) \
	do { \
		hipError_t cudaCheckReturn_e = (ret); \
		if (cudaCheckReturn_e != hipSuccess) { \
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
			fflush(stderr); \
		} \
		assert(cudaCheckReturn_e == hipSuccess); \
	} while(0)

#define cudaCheckKernel() \
	do { \
		cudaCheckReturn(hipGetLastError()); \
	} while(0)

void init_array(DATA_TYPE *A, DATA_TYPE *p, DATA_TYPE *r)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		r[i] = i * M_PI;

		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*j) / NX;
		}
	}
	
	for (i = 0; i < NY; i++)
	{
		p[i] = i * M_PI;
	}
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	cudaCheckReturn(hipGetDeviceProperties(&deviceProp, GPU_DEVICE));
	cudaCheckReturn(hipSetDevice(GPU_DEVICE));
}

//Distributed (split) from initial loop and permuted into reverse order to allow parallelism...
__global__ void bicg_kernel1(DATA_TYPE *A, DATA_TYPE *r, DATA_TYPE *s)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < NY)
	{
		s[j] = 0.0f;

		int i;
		for(i = 0; i < NX; i++)
		{
			s[j] += A[i * NY + j] * r[i];
		}
	}	
}

//Distributed (split) from initial loop to allow parallelism
__global__ void bicg_kernel2(DATA_TYPE *A, DATA_TYPE *p, DATA_TYPE *q)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < NX)
	{
		q[i] = 0.0f;

		int j;
		for(j = 0; j < NY; j++)
		{
			q[i] += A[i * NY + j] * p[j];
		}
	}
}

void bicgCuda(DATA_TYPE* A, DATA_TYPE* r, DATA_TYPE* s, DATA_TYPE* p, DATA_TYPE* q)
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *q_gpu;
	DATA_TYPE *p_gpu;
	DATA_TYPE *r_gpu;
	DATA_TYPE *s_gpu;

	cudaCheckReturn(hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY));
	cudaCheckReturn(hipMalloc((void **)&r_gpu, sizeof(DATA_TYPE) * NX));
	cudaCheckReturn(hipMalloc((void **)&s_gpu, sizeof(DATA_TYPE) * NY));
	cudaCheckReturn(hipMalloc((void **)&p_gpu, sizeof(DATA_TYPE) * NY));
	cudaCheckReturn(hipMalloc((void **)&q_gpu, sizeof(DATA_TYPE) * NX));
	cudaCheckReturn(hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(r_gpu, r, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(s_gpu, s, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(p_gpu, p, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice));
	cudaCheckReturn(hipMemcpy(q_gpu, q, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice));

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);

	bicg_kernel1<<< grid1, block >>>(A_gpu, r_gpu, s_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	bicg_kernel2<<< grid2, block >>>(A_gpu, p_gpu, q_gpu);
	cudaCheckKernel();
	cudaCheckReturn(hipDeviceSynchronize());
	
	cudaCheckReturn(hipMemcpy(s, s_gpu, sizeof(DATA_TYPE) * NY, hipMemcpyDeviceToHost));
	cudaCheckReturn(hipMemcpy(q, q_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost));

	cudaCheckReturn(hipFree(A_gpu));
	cudaCheckReturn(hipFree(r_gpu));
	cudaCheckReturn(hipFree(s_gpu));
	cudaCheckReturn(hipFree(p_gpu));
	cudaCheckReturn(hipFree(q_gpu));
}

int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* r;
	DATA_TYPE* s;
	DATA_TYPE* p;
	DATA_TYPE* q;
 	
	A = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
	r = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));
	s = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	p = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	q = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));

	init_array(A, p, r);

	GPU_argv_init();

	t_start = rtclock();
	bicgCuda(A, r, s, p, q);
	t_end = rtclock();

#ifdef POLYBENCH_TIME
	fprintf(stdout, "%0.6lf\n", t_end - t_start);
#endif

	free(A);
	free(r);
	free(s);
	free(p);
	free(q);

  	return 0;
}

